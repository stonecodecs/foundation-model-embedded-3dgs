/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include <fstream>
#include <string>
#include <functional>

// Rocky: creates and returns a callable object (lambda) that can be used to resize a PyTorch tensor and obtain a pointer to its data.
// This can be helpful when you want to manage memory and perform operations on the tensor's data.
// The returned callable object takes input of (size_t N), and returns char*.
std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::tuple<int, torch::Tensor, torch::Tensor,  torch::Tensor,  torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors_precomp,
    const torch::Tensor& opacity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx,
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const bool prefiltered,
	const bool debug)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }

  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;
  const int FEATURES_CH = colors_precomp.size(1);
  const int FEATURES_EX_CH = VL_FEATURE_EX_NUM_CHANNELS;  // Use the value from config.h
  
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kFloat32);

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor out_color = torch::full({FEATURES_CH, H, W}, 0.0, float_opts);
  torch::Tensor out_color_ex = torch::full({FEATURES_EX_CH, H, W}, 0.0, float_opts);  // Create with proper dimensions
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));

  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);

  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
      }

	  rendered = CudaRasterizer::Rasterizer::forward(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P, degree, M,
		background.contiguous().data<float>(),
		W, H, FEATURES_CH, FEATURES_EX_CH,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(),
        colors_precomp.contiguous().data<float>(),
        nullptr,  // Pass nullptr for the second feature map's data
		opacity.contiguous().data<float>(),
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		out_color.contiguous().data<float>(),
        out_color_ex.contiguous().data<float>(),  // Pass the output tensor
		radii.contiguous().data<int>(),
		debug);
  }
  return std::make_tuple(rendered, out_color, out_color_ex, radii, geomBuffer, binningBuffer, imgBuffer);
}

torch::Tensor
RasterizeGaussiansBackwardCUDA(
 	const torch::Tensor& background,
	const torch::Tensor& means3D,
	const torch::Tensor& radii,
    const torch::Tensor& colors_precomp,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
	const float tan_fovx,
	const float tan_fovy,
    const torch::Tensor& dL_dout_color,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const torch::Tensor& geomBuffer,
	const int R,
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const bool debug)
{
  const int P = means3D.size(0);
  const int H = dL_dout_color.size(1);
  const int W = dL_dout_color.size(2);
  const int FEATURES_CH = colors_precomp.size(1);
  const int FEATURES_EX_CH = VL_FEATURE_EX_NUM_CHANNELS;  // Use the value from config.h

  int M = 0;
  if(sh.size(0) != 0)
  {
	M = sh.size(1);
  }

  const int num_channels_colors_precomp = colors_precomp.size(1);
  torch::Tensor dL_dcolorsprecom = torch::zeros({P, num_channels_colors_precomp}, means3D.options());

  if(P != 0)
  {
	  CudaRasterizer::Rasterizer::backward(P, degree, M, R,
	  background.contiguous().data<float>(),
	  W, H, FEATURES_CH, FEATURES_EX_CH,
	  means3D.contiguous().data<float>(),
	  sh.contiguous().data<float>(),
      colors_precomp.contiguous().data<float>(),
      nullptr,  // Pass nullptr for the second feature map's data
	  scales.data_ptr<float>(),
	  scale_modifier,
	  rotations.data_ptr<float>(),
	  cov3D_precomp.contiguous().data<float>(),
	  viewmatrix.contiguous().data<float>(),
	  projmatrix.contiguous().data<float>(),
	  campos.contiguous().data<float>(),
	  tan_fovx,
	  tan_fovy,
	  radii.contiguous().data<int>(),
	  reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
	  dL_dout_color.contiguous().data<float>(),
      nullptr,  // Pass nullptr for the second feature map's gradient
	  dL_dcolorsprecom.contiguous().data<float>(),
      nullptr,  // Pass nullptr for the second feature map's gradient
	  debug);
  }

  return dL_dcolorsprecom;
}

torch::Tensor markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix,
		torch::Tensor& projmatrix)
{
  const int P = means3D.size(0);

  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));

  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		projmatrix.contiguous().data<float>(),
		present.contiguous().data<bool>());
  }

  return present;
}