/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include <fstream>
#include <string>
#include <functional>

// Rocky: creates and returns a callable object (lambda) that can be used to resize a PyTorch tensor and obtain a pointer to its data.
// This can be helpful when you want to manage memory and perform operations on the tensor's data.
// The returned callable object takes input of (size_t N), and returns char*.
std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::tuple<int, torch::Tensor, torch::Tensor,  torch::Tensor,  torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors_precomp,
    const torch::Tensor& colors_ex_precomp,
    const torch::Tensor& opacity,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx, 
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const bool prefiltered,
	const bool debug)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }
  
  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;
//   printf("Before accessing size(1)\n");
  const int FEATURES_CH = colors_precomp.size(1);
  // TODO: remove features_ex_ch later from codebase
  const int FEATURES_EX_CH = colors_ex_precomp.size(1);
//   printf("After accessing size(1)\n");
  //printf("FEATURES_CH, VL_FEATURE_NUM_CHANNELS in RasterizeGaussiansCUDA(): %d, %d\n", FEATURES_CH, VL_FEATURE_NUM_CHANNELS);

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor out_color = torch::full({FEATURES_CH, H, W}, 0.0, float_opts); // This can be feature map or RGB image.
  torch::Tensor out_color_ex = torch::full({FEATURES_EX_CH, H, W}, 0.0, float_opts); // This can be feature map or RGB image.
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
  
  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
      }

	  rendered = CudaRasterizer::Rasterizer::forward(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P, degree, M,
		background.contiguous().data<float>(),
		W, H, FEATURES_CH, FEATURES_EX_CH,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(),
        colors_precomp.contiguous().data<float>(),
        colors_ex_precomp.contiguous().data<float>(),
		opacity.contiguous().data<float>(), 
		scales.contiguous().data_ptr<float>(),
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		out_color.contiguous().data<float>(),
        out_color_ex.contiguous().data<float>(),
		radii.contiguous().data<int>(),
		debug);
  }
  return std::make_tuple(rendered, out_color, out_color_ex, radii, geomBuffer, binningBuffer, imgBuffer);
}

//std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
std::tuple<torch::Tensor, torch::Tensor>
RasterizeGaussiansBackwardCUDA(
 	const torch::Tensor& background,
	const torch::Tensor& means3D,
	const torch::Tensor& radii,
    const torch::Tensor& colors_precomp,
    const torch::Tensor& colors_ex_precomp,
	const torch::Tensor& scales,
	const torch::Tensor& rotations,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
	const float tan_fovx,
	const float tan_fovy,
    const torch::Tensor& dL_dout_color,
    const torch::Tensor& dL_dout_color_ex,
	const torch::Tensor& sh,
	const int degree,
	const torch::Tensor& campos,
	const torch::Tensor& geomBuffer,
	const int R,
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const bool debug) 
{
  const int P = means3D.size(0);
  const int H = dL_dout_color.size(1);
  const int W = dL_dout_color.size(2);
  const int FEATURES_CH = colors_precomp.size(1);
  const int FEATURES_EX_CH = colors_ex_precomp.size(1);
  
  int M = 0;
  if(sh.size(0) != 0)
  {	
	M = sh.size(1);
  }

  const int num_channels_colors_precomp = colors_precomp.size(1);
  torch::Tensor dL_dcolorsprecom = torch::zeros({P, num_channels_colors_precomp}, means3D.options());
  const int num_channels_colors_ex_precomp = colors_ex_precomp.size(1);
  torch::Tensor dL_dcolorsprecom_ex = torch::zeros({P, num_channels_colors_ex_precomp}, means3D.options());

  
  if(P != 0)
  {  
	  CudaRasterizer::Rasterizer::backward(P, degree, M, R,
	  background.contiguous().data<float>(),
	  W, H, FEATURES_CH, FEATURES_EX_CH,
	  means3D.contiguous().data<float>(),
	  sh.contiguous().data<float>(),
      colors_precomp.contiguous().data<float>(),
      colors_ex_precomp.contiguous().data<float>(),
	  scales.data_ptr<float>(),
	  scale_modifier,
	  rotations.data_ptr<float>(),
	  cov3D_precomp.contiguous().data<float>(),
	  viewmatrix.contiguous().data<float>(),
	  projmatrix.contiguous().data<float>(),
	  campos.contiguous().data<float>(),
	  tan_fovx,
	  tan_fovy,
	  radii.contiguous().data<int>(),
	  reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
	  dL_dout_color.contiguous().data<float>(),
      dL_dout_color_ex.contiguous().data<float>(),
	  dL_dcolorsprecom.contiguous().data<float>(), // Gradient of loss w.r.t. colors_precomp
      dL_dcolorsprecom_ex.contiguous().data<float>(), // Gradient of loss w.r.t. colors_precomp
	  debug);
  }

//  return std::make_tuple(dL_dmeans2D, dL_dcolorsprecom, dL_dopacity, dL_dmeans3D, dL_dcov3D, dL_dsh, dL_dscales, dL_drotations);
  return std::make_tuple(dL_dcolorsprecom, dL_dcolorsprecom_ex);
}

torch::Tensor markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix,
		torch::Tensor& projmatrix)
{ 
  const int P = means3D.size(0);
  
  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
 
  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		projmatrix.contiguous().data<float>(),
		present.contiguous().data<bool>());
  }
  
  return present;
}